#include <hip/hip_runtime.h>
#include <stdio.h>
#include "../common/common.h"

/** 
 * 一个使用 CUDA 内存拷贝 API 在设备和主机之间传输数据的示例。
 * 
 * 在这个示例中，使用 hipMalloc 在 GPU 上分配内存，
 * 并使用 hipMemcpy 将主机内存中的内容传输到通过 hipMalloc 分配的数组中。
 * 主机内存通过 hipHostMalloc 分配，以创建一个页锁定（page-locked）的主机数组。
 */

int main(int argc, char **argv)
{
    // set up device
    int dev = getGPUId();
    CHECK(hipSetDevice(dev));

    // memory size
    unsigned int isize  = 1 << 22;
    unsigned int nbytes = isize * sizeof(float);

    // get device information
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));

    if (!deviceProp.canMapHostMemory)
    {
        printf("Device %d does not support mapping CPU host memory!\n", dev);
        CHECK(hipDeviceReset());
        exit(EXIT_SUCCESS);
    }

    printf("%s starting at ", argv[0]);
    printf("device %d: %s memory size %d nbyte %5.2fMB, canMap = %d\n", dev, deviceProp.name, isize,
           nbytes / (1024.0f * 1024.0f), deviceProp.canMapHostMemory);

    // allocate pinned host memory
    float *h_a;
    CHECK(hipHostMalloc((float **)&h_a, nbytes));

    // allocate device memory
    float *d_a;
    CHECK(hipMalloc((float **)&d_a, nbytes));

    // initialize host memory
    memset(h_a, 0, nbytes);

    for (int i = 0; i < isize; i++)
        h_a[i] = 100.10f;

    // transfer data from the host to the device
    CHECK(hipMemcpy(d_a, h_a, nbytes, hipMemcpyHostToDevice));

    // transfer data from the device to the host
    CHECK(hipMemcpy(h_a, d_a, nbytes, hipMemcpyDeviceToHost));

    // free memory
    CHECK(hipFree(d_a));
    CHECK(hipHostFree(h_a));

    // reset device
    CHECK(hipDeviceReset());
    return EXIT_SUCCESS;
}
