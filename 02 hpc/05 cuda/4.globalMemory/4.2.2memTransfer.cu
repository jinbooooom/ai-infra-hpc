#include <hip/hip_runtime.h>
#include <stdio.h>
#include "../common/common.h"

/*
 * 一个使用 CUDA 内存拷贝 API 在设备和主机之间传输数据的示例。
 * 在这个示例中，使用 hipMalloc 在 GPU 上分配内存，
 * 并使用 hipMemcpy 将主机内存中的内容传输到通过 hipMalloc 分配的数组中。
 */

int main(int argc, char **argv)
{
    // set up device
    int dev = getGPUId();
    CHECK(hipSetDevice(dev));

    // memory size
    unsigned int isize  = 1 << 22;
    unsigned int nbytes = isize * sizeof(float);

    // get device information
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("%s starting at ", argv[0]);
    printf("device %d: %s memory size %d nbyte %5.2fMB\n", dev, deviceProp.name, isize, nbytes / (1024.0f * 1024.0f));

    // allocate the host memory
    float *h_a = (float *)malloc(nbytes);

    // allocate the device memory
    float *d_a;
    CHECK(hipMalloc((float **)&d_a, nbytes));

    // initialize the host memory
    for (unsigned int i = 0; i < isize; i++)
        h_a[i] = 0.5f;

    // transfer data from the host to the device
    CHECK(hipMemcpy(d_a, h_a, nbytes, hipMemcpyHostToDevice));

    // transfer data from the device to the host
    CHECK(hipMemcpy(h_a, d_a, nbytes, hipMemcpyDeviceToHost));

    // free memory
    CHECK(hipFree(d_a));
    free(h_a);

    // reset device
    CHECK(hipDeviceReset());
    return EXIT_SUCCESS;
}
