#include <hip/hip_runtime.h>
#include <stdio.h>
#include "../common/common.h"

/*
 * 获取当前 CUDA 平台上第一个设备的基本信息，
 * 包括流多处理器（SM）的数量、常量内存的字节数、
 * 每个块的共享内存字节数等。
 */

int main(int argc, char *argv[])
{
    int iDev = 0;
    hipDeviceProp_t iProp;
    CHECK(hipGetDeviceProperties(&iProp, iDev));

    printf("Device %d: %s\n", iDev, iProp.name);
    printf("  Number of multiprocessors:                     %d\n", iProp.multiProcessorCount);
    printf("  Total amount of constant memory:               %4.2f KB\n", iProp.totalConstMem / 1024.0);
    printf("  Total amount of shared memory per block:       %4.2f KB\n", iProp.sharedMemPerBlock / 1024.0);
    printf("  Total number of registers available per block: %d\n", iProp.regsPerBlock);
    printf("  Warp size:                                     %d\n", iProp.warpSize);
    printf("  Maximum number of threads per block:           %d\n", iProp.maxThreadsPerBlock);
    printf("  Maximum number of threads per multiprocessor:  %d\n", iProp.maxThreadsPerMultiProcessor);
    printf("  Maximum number of warps per multiprocessor:    %d\n", iProp.maxThreadsPerMultiProcessor / 32);
    return EXIT_SUCCESS;
}
